#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_blockArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<VolumeType, 3, hipReadModeNormalizedFloat> tex_block;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}
__device__ unsigned char myMAX(unsigned char a, unsigned char b)
{
	if(a >= b)
		return a;
	else 
		return b;
}
__device__ 
float3 cudaNormalize(float3 a){
	float3 temp={a.x, a.y, a.z};
	float sum = sqrt((float)(a.x*a.x + a.y*a.y + a.z*a.z));

	if(sum == 0){
		temp.x = 0;
		temp.y = 0;
		temp.z = 0;
	}else{
		temp.x /= sum;
		temp.y /= sum;
		temp.z /= sum;
	}

	return temp;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}
__device__ uchar rgbaFloatToChar(float rgba)
{
	rgba = __saturatef(rgba);   // clamp to [0.0, 1.0]
	return (uchar(rgba*255));
}
__global__ void makeBlock_kernel(unsigned char* image_p, unsigned char* dest_p, hipExtent blockSize, hipExtent volumeSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (tx >= blockSize.width || ty >= blockSize.height) return;

	for(int i=0; i<blockSize.depth; i++){
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = 0;
		unsigned char tempmax=0;

		for(int z=i*4; z<=i*4+4; z++)
			for(int y=ty*4; y<=ty*4+4; y++)
				for(int x=tx*4; x<=tx*4+4; x++){
					if(z>=volumeSize.depth || y>=volumeSize.height || x>=volumeSize.width )
						continue;
					tempmax = myMAX(tempmax, image_p[z*volumeSize.width*volumeSize.height + y*volumeSize.height + x]);
				}
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = tempmax;
	}
}
__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
	float4 temp =make_float4(0.0f);
	//uint4 sum = make_uint4(0);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d * tnear;
    float3 step = eyeRay.d*tstep;
	float max = 0.0f; 
    for (float i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates

	   // float block_den = tex3D(tex_block, (pos.x*0.5f+0.5f), (pos.y*0.5f+0.5f), (pos.z*0.5f+0.5f));
		//uint density = __float2uint_rn(block_den*256);
		/*temp.w = block_den;
		temp.x = block_den;
		temp.y = block_den;
		temp.z = block_den;
		uint density =  ((unsigned int)(temp.w*255)<<24) | ((unsigned int)(temp.z*255)<<16) | ((unsigned int)(temp.y*255)<<8) | (unsigned int)(temp.x*255);*/
	//	if(block_den >= max) 
//				max = block_den;*/
		//if(((density >> 16) &255) < 4) { //����� ���� - PALLET_START~PALLET_END������ �׸��� ������
		//	
		//}
		//else{
			float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
	       
			// lookup in transfer function texture
			float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
	      
			float3 nV = {0.0, 0.0, 0.0};
			float3 lV = {0.0, 0.0, 0.0};

			lV.x = eyeRay.d.x;
			lV.y = eyeRay.d.y;
			lV.z = eyeRay.d.z;
			
			float x_plus = tex3D(tex, pos.x*0.5f+0.5+(step.x*0.5), pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
			float x_minus = tex3D(tex,pos.x*0.5f+0.5-(step.x*0.5), pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

			float y_plus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f +(step.y*0.5), pos.z*0.5f+0.5f);
			float y_minus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f-(step.y*0.5),pos.z*0.5f+0.5f);

			float z_plus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f+(step.z*0.5));
			float z_minus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f-(step.z*0.5));

			nV.x = (x_plus - x_minus)/2.0f;
			nV.y = (y_plus - y_minus)/2.0f;
			nV.z = (z_plus - z_minus)/2.0f;

			//nV = cudaNormalize(nV);

			float NL = 0.0f;
			NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;

			if(NL < 0.0f) NL = 0.0f;
			float localShading = 0.2 + 0.8*NL;
			
			//col*=localShading;
			// pre-multiply alpha
			col.x *= col.w;
			col.y *= col.w;
			col.z *= col.w;
			// "over" operator for front-to-back blending
			sum = sum + col*(1.0f - sum.w);

			// exit early if opaque
			if (sum.w > opacityThreshold)
				break;

			t += (tstep*0.5);

			if (t > tfar) break;

			pos += (step*0.5);
		//}
	}
	/*sum.x = max;
	sum.y = max;
	sum.z = max;
	sum.w = 0;*/
    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
	
}

extern "C"
void* make_blockVolume(void* image, hipExtent blockSize, hipExtent volumeSize)
{
	unsigned int vsize = volumeSize.width * volumeSize.height * volumeSize.depth * sizeof(VolumeType);
	unsigned int bsize = blockSize.width * blockSize.height * blockSize.depth * sizeof(VolumeType);

	unsigned char *dest; //cpu�� ���� ��� data
	unsigned char *dest_p; //gpu���� ����� ��� ������
	unsigned char *image_p; //���� ������

	dest = new unsigned char[bsize/sizeof(VolumeType)]; //64*64*57
	memset((void*)dest, 0, bsize);

	hipMalloc((void**)&image_p, vsize); 
	hipMemcpy(image_p, image, vsize, hipMemcpyHostToDevice); 

	hipMalloc((void**)&dest_p, bsize);

	dim3 Db = dim3(16, 16);
	dim3 Dg = dim3(4, 4);

	makeBlock_kernel<<<Dg, Db>>>(image_p, dest_p, blockSize, volumeSize);

	hipMemcpy(dest, dest_p, bsize, hipMemcpyDeviceToHost);
	/*for(int i=0; i<64*64*47; i++)
	{
		printf("%d\n",dest[i]);
	}*/
	hipFree(image_p);
	hipFree(dest_p);

	return dest;

}
extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}
extern "C"
void initBlockTexture(void *h_volume_block, int x, int y, int z)
{
	hipExtent block_Size = make_hipExtent(x, y, z);
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors( hipMalloc3DArray(&d_blockArray, &channelDesc, block_Size) );

    // copy data to 3D array
    hipMemcpy3DParms myParams = {0};
	myParams.srcPtr   = make_hipPitchedPtr(h_volume_block, block_Size.width*sizeof(VolumeType), block_Size.width, block_Size.height);
    myParams.dstArray = d_blockArray;
    myParams.extent   = block_Size;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_block.normalized = true;                      // access with normalized texture coordinates
    tex_block.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_block.channelDesc = channelDesc;
	tex_block.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates
    tex_block.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_block, d_blockArray, channelDesc));            
} 
extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeBorder;
    // tex.addressMode[2] = hipAddressModeBorder;
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

	float4 transferFunc[256];
   
	 for(int i=0; i<=80; i++){    //alpha
		 transferFunc[i].w = 0.0f;
		 transferFunc[i].x = 0.0f;
		 transferFunc[i].y = 0.0f;
		 transferFunc[i].z = 0.0f;
	}
	for(int i=80+1; i<=100; i++){
		transferFunc[i].w = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].x = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].y = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].z = (1.0 / (100-80)) * ( i - 80);
	}
	for(int i=100+1; i<256; i++){
		transferFunc[i].w =1.0f;
		transferFunc[i].x =1.0f;
		transferFunc[i].y =1.0f;
		transferFunc[i].z =1.0f;
	}
	//-------------------------------------------------------------------
	// create transfer function texture
  //  float4 transferFunc[] =
  //  {
  //     /* {  0.0, 0.0, 0.0, 0.0, },
  //      {  1.0, 0.0, 0.0, 1.0, },
  //      {  1.0, 0.5, 0.0, 1.0, },
  //      {  1.0, 1.0, 0.0, 1.0, },
  //      {  0.0, 1.0, 0.0, 1.0, },
  //      {  0.0, 1.0, 1.0, 1.0, },
  //      {  0.0, 0.0, 1.0, 1.0, },
  //      {  1.0, 0.0, 1.0, 1.0, },
  //      {  0.0, 0.0, 0.0, 0.0, },*/

		//{  0.0, 0.0, 0.0, 0.0, },
  //      {  0.0, 0.0, 0.0, 1.0, },
  //      {  0.0, 0.0, 0.1, 0.2, },
  //      {  0.3, 0.4, 0.5, 0.6, },
  //      {  0.7, 0.8, 0.9, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 0.0, },
  //  };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                      brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
