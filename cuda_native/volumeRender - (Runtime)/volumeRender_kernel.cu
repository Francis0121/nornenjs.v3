#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_blockArray = 0;
hipArray *d_transferFuncArray;
hipArray *d_transferFuncArray1 = 0;
typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<VolumeType, 3, hipReadModeNormalizedFloat> tex_block;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture
texture<float4, 1, hipReadModeElementType>         transferTex1; // 1D transfer function texture
typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}
__device__ unsigned char myMAX(unsigned char a, unsigned char b)
{
	if(a >= b)
		return a;
	else 
		return b;
}
__device__ 
float3 cudaNormalize(float3 a){
	float3 temp={a.x, a.y, a.z};
	float sum = sqrt((float)(a.x*a.x + a.y*a.y + a.z*a.z));

	if(sum == 0){
		temp.x = 0;
		temp.y = 0;
		temp.z = 0;
	}else{
		temp.x /= sum;
		temp.y /= sum;
		temp.z /= sum;
	}

	return temp;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}
__device__ uchar rgbaFloatToChar(float rgba)
{
	rgba = __saturatef(rgba);   // clamp to [0.0, 1.0]
	return (uchar(rgba*255));
}
__global__ void makeBlock_kernel(unsigned char* image_p, unsigned char* dest_p, hipExtent blockSize, hipExtent volumeSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (tx >= blockSize.width || ty >= blockSize.height) return;

	for(int i=0; i<blockSize.depth; i++){
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = 0;
		unsigned char tempmax=0;

		for(int z=i*4; z<=i*4+4; z++)
			for(int y=ty*4; y<=ty*4+4; y++)
				for(int x=tx*4; x<=tx*4+4; x++){
					if(z>=volumeSize.depth || y>=volumeSize.height || x>=volumeSize.width )
						continue;
					tempmax = myMAX(tempmax, image_p[z*volumeSize.width*volumeSize.height + y*volumeSize.height + x]);
				}
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = tempmax;
	}
}
__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
	float4 temp =make_float4(0.0f);
	//uint4 sum = make_uint4(0);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d * tnear;
    float3 step = eyeRay.d*tstep;
	float max = 0.0f; 
    for (float i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates

	   // float block_den = tex3D(tex_block, (pos.x*0.5f+0.5f), (pos.y*0.5f+0.5f), (pos.z*0.5f+0.5f))*65535;
		//float3 advanced = {0.0f,0.0f.0.0f};
		//uint density = __float2uint_rn(block_den*256);
		/*temp.w = block_den;
		temp.x = block_den;
		temp.y = block_den;
		temp.z = block_den;
		uint density =  ((unsigned int)(temp.w*255)<<24) | ((unsigned int)(temp.z*255)<<16) | ((unsigned int)(temp.y*255)<<8) | (unsigned int)(temp.x*255);*/
	   //	if(block_den >= max) 
       //				max = block_den;*/
	   //if((int)block_den < 80) { //����� ���� - PALLET_START~PALLET_END������ �׸��� ������
		  // int3 nowPos= {(pos.x*0.5f+0.5f), (pos.y*0.5f+0.5f), (pos.z*0.5f+0.5f)};
		  // int3 advpos;
		  // do{
				//pos += (step*0.5);
			
		  // }
		
	    //
	    //}
		//else{
			float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
	        float sample_next = tex3D(tex, pos.x*0.5f+0.5+(step.x*0.5), pos.y*0.5f+0.5f +(step.y*0.5), pos.z*0.5f+0.5f+(step.z*0.5));
			
			// lookup in transfer function texture
			//float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
			//float4 col={0.0};
			float diff;
			//if(sample<=sample_next){
				 diff=sample_next-sample;
				 float4 col= (tex1D(transferTex1, (sample_next-transferOffset)*transferScale) - tex1D(transferTex1, (sample-transferOffset)*transferScale)) / diff;
				
			//}
			//else if(sample>sample_next){
			//	diff=sample-sample_next;
			///	col= (tex1D(transferTex1, (sample-transferOffset)*transferScale) - tex1D(transferTex1, (sample_next-transferOffset)*transferScale)) / diff;
			//}
			//float4 col = tex3D(transferTex1,sample,sample_next,0);


			//float3 nV = {0.0, 0.0, 0.0};
			//float3 lV = {0.0, 0.0, 0.0};

			//lV.x = eyeRay.d.x;
			//lV.y = eyeRay.d.y;
			//lV.z = eyeRay.d.z;
			//
			//float x_plus = tex3D(tex, pos.x*0.5f+0.5+(step.x*0.5), pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
			//float x_minus = tex3D(tex,pos.x*0.5f+0.5-(step.x*0.5), pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

			//float y_plus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f +(step.y*0.5), pos.z*0.5f+0.5f);
			//float y_minus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f-(step.y*0.5),pos.z*0.5f+0.5f);

			//float z_plus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f+(step.z*0.5));
			//float z_minus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f-(step.z*0.5));

			//nV.x = (x_plus - x_minus)/2.0f;
			//nV.y = (y_plus - y_minus)/2.0f;
			//nV.z = (z_plus - z_minus)/2.0f;

			////nV = cudaNormalize(nV);

			//float NL = 0.0f;
			//NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;

			//if(NL < 0.0f) NL = 0.0f;
			//float localShading = 0.2 + 0.8*NL;
			
			//col*=localShading;
			// pre-multiply alpha
			col.x *= col.w;
			col.y *= col.w;
			col.z *= col.w;
			// "over" operator for front-to-back blending
			sum = sum + col*(1.0f - sum.w);

			// exit early if opaque
			if (sum.w > opacityThreshold)
				break;

			t += (tstep*0.5);

			if (t > tfar) break;

			pos += (step*0.5);
		//}
	}
	/*sum.x = max;
	sum.y = max;
	sum.z = max;
	sum.w = 0;*/
    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
	
}

extern "C"
void* make_blockVolume(void* image, hipExtent blockSize, hipExtent volumeSize)
{
	unsigned int vsize = volumeSize.width * volumeSize.height * volumeSize.depth * sizeof(VolumeType);
	unsigned int bsize = blockSize.width * blockSize.height * blockSize.depth * sizeof(VolumeType);

	unsigned char *dest; //cpu�� ���� ��� data
	unsigned char *dest_p; //gpu���� ����� ��� ������
	unsigned char *image_p; //���� ������

	dest = new unsigned char[bsize/sizeof(VolumeType)]; //64*64*57
	memset((void*)dest, 0, bsize);

	hipMalloc((void**)&image_p, vsize); 
	hipMemcpy(image_p, image, vsize, hipMemcpyHostToDevice); 

	hipMalloc((void**)&dest_p, bsize);

	dim3 Db = dim3(16, 16);
	dim3 Dg = dim3(4, 4);

	makeBlock_kernel<<<Dg, Db>>>(image_p, dest_p, blockSize, volumeSize);

	hipMemcpy(dest, dest_p, bsize, hipMemcpyDeviceToHost);
	/*for(int i=0; i<64*64*47; i++)
	{
		printf("%d\n",dest[i]);
	}*/
	hipFree(image_p);
	hipFree(dest_p);

	return dest;

}
extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}
extern "C"
void initBlockTexture(void *h_volume_block, int x, int y, int z)
{
	hipExtent block_Size = make_hipExtent(x, y, z);
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors( hipMalloc3DArray(&d_blockArray, &channelDesc, block_Size) );

    // copy data to 3D array
    hipMemcpy3DParms myParams = {0};
	myParams.srcPtr   = make_hipPitchedPtr(h_volume_block, block_Size.width*sizeof(VolumeType), block_Size.width, block_Size.height);
    myParams.dstArray = d_blockArray;
    myParams.extent   = block_Size;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_block.normalized = true;                      // access with normalized texture coordinates
    tex_block.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_block.channelDesc = channelDesc;
	tex_block.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates
    tex_block.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_block, d_blockArray, channelDesc));            
} 




//struct OTF_2D* getPre_integration(){
//
//	
//	for(int x=0; x<256; x++){
//		for(int y=0; y<256; y++){
//
//			float4 result;
//			float4 temp={0.0f};
//
//			if(y > x){
//				for(int i=x; i<y; i++){
//					temp.x = transferFunc[i].x;
//					temp.y = transferFunc[i].y;
//					temp.z = transferFunc[i].z;
//					temp.w = transferFunc[i].w;
//					
//					float diff = i-x;
//
//					if(diff == 0.0f)
//						diff = 1.0f;
//
//					temp.w = 1.0f-pow(1-temp.w, 1/diff);
//
//					result.x += (1-result.w)*temp.x*temp.w;
//					result.y += (1-result.w)*temp.y*temp.w;
//					result.z += (1-result.w)*temp.z*temp.w;
//					result.w += (1-result.w)*temp.w;
//				}
//			}
//			else if(x > y){
//				for(int i=y; i<x; i++){
//					temp.x = transferFunc[i].x;
//					temp.y = transferFunc[i].y;
//					temp.z = transferFunc[i].z;
//					temp.w = transferFunc[i].w;
//
//					float diff = i-y;
//
//					if(diff == 0.0f)
//						diff = 1.0f;
//
//					temp.w = 1.0f-pow(1-temp.w, 1/diff);
//
//					result.x += (1-result.w)*temp.x*temp.w;
//					result.y += (1-result.w)*temp.y*temp.w;
//					result.z += (1-result.w)*temp.z*temp.w;
//					result.w += (1-result.w)*temp.w;
//				}
//			}
//			else {
//				result.x = 255.0f;
//				result.y = 255.0f;
//				result.z = 255.0f;
//				result.w = 0.0f;
//			}
//			OTF_2D[256*x + y].sum_R = result.x;
//			OTF_2D[256*x + y].sum_G = result.y;
//			OTF_2D[256*x + y].sum_B = result.z;
//			OTF_2D[256*x + y].sum_a = result.w;
//		}
//	}
//	return OTF_2D;
//}
extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeBorder;
    // tex.addressMode[2] = hipAddressModeBorder;
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

		
	float4 transferFunc[256];
	float4 transferFunc1[256]={0.0f};
	 for(int i=0; i<=80; i++){    //alpha
		 transferFunc[i].w = 0.0f;
		 transferFunc[i].x = 0.0f;
		 transferFunc[i].y = 0.0f;
		 transferFunc[i].z = 0.0f;
	}
	for(int i=80+1; i<=100; i++){
		transferFunc[i].w = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].x = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].y = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].z = (1.0 / (100-80)) * ( i - 80);
	}
	for(int i=100+1; i<256; i++){
		transferFunc[i].w =1.0f;
		transferFunc[i].x =1.0f;
		transferFunc[i].y =1.0f;
		transferFunc[i].z =1.0f;
	}
	
	transferFunc1[0].w= transferFunc[0].w;
	transferFunc1[0].x= transferFunc[0].x * transferFunc[0].w;
	transferFunc1[0].y= transferFunc[0].y * transferFunc[0].w;
	transferFunc1[0].z= transferFunc[0].z * transferFunc[0].w;		
	
	for(int i=1; i<256; i++)
	{		
		

		transferFunc1[i].w += transferFunc1[i-1].w + transferFunc[i].w;
		transferFunc1[i].x += transferFunc1[i-1].x + transferFunc[i].x * transferFunc[i].w;
		transferFunc1[i].y += transferFunc1[i-1].y + transferFunc[i].y * transferFunc[i].w;
		transferFunc1[i].z += transferFunc1[i-1].z + transferFunc[i].z * transferFunc[i].w;
		
		transferFunc1[i].w =(transferFunc1[i].w/256.0f);
		transferFunc1[i].x =(transferFunc1[i].x/256.0f);
		transferFunc1[i].y =(transferFunc1[i].y/256.0f);
		transferFunc1[i].z =(transferFunc1[i].z/256.0f);
		//printf("%f %f\n",transferFunc1[i].w/256,transferFunc1[i].x/256);
		//printf("%f,%f,%f,%f\n",tempA[i],OTF_2Da[before],tempG[i],OTF_2Dg[before]);

	}
	//for(int x=0; x<256; x++){
	//	for(int y=0; y<256; y++){

	//		float4 result;
	//		float4 temp={0.0f};

	//		if(y > x){
	//			for(int i=x; i<y; i++){
	//				temp.x = transferFunc[i].x;
	//				temp.y = transferFunc[i].y;
	//				temp.z = transferFunc[i].z;
	//				temp.w = transferFunc[i].w;
	//				
	//				float diff = i-x;

	//				if(diff == 0.0f)
	//					diff = 1.0f;

	//				temp.w = 1.0f-pow(1-temp.w, 1/diff);

	//				result.x += (1-result.w)*temp.x*temp.w;
	//				result.y += (1-result.w)*temp.y*temp.w;
	//				result.z += (1-result.w)*temp.z*temp.w;
	//				result.w += (1-result.w)*temp.w;
	//			}
	//		}
	//		else if(x > y){
	//			for(int i=y; i<x; i++){
	//				temp.x = transferFunc[i].x;
	//				temp.y = transferFunc[i].y;
	//				temp.z = transferFunc[i].z;
	//				temp.w = transferFunc[i].w;

	//				float diff = i-y;

	//				if(diff == 0.0f)
	//					diff = 1.0f;

	//				temp.w = 1.0f-pow(1-temp.w, 1/diff);

	//				result.x += (1-result.w)*temp.x*temp.w;
	//				result.y += (1-result.w)*temp.y*temp.w;
	//				result.z += (1-result.w)*temp.z*temp.w;
	//				result.w += (1-result.w)*temp.w;
	//			}
	//		}
	//		else {
	//			result.x = 1.0f;
	//			result.y = 1.0f;
	//			result.z = 1.0f;
	//			result.w = 0.0f;
	//		}
	//		OTF_2D[256*x + y].sum_R = result.x;
	//		OTF_2D[256*x + y].sum_G = result.y;
	//		OTF_2D[256*x + y].sum_B = result.z;
	//		OTF_2D[256*x + y].sum_a = result.w;
	//	}
	//}
	//struct OTF_2D *p;
	//p=getPre_integration();
	//for(int i=0; i<256; i++)
	//{
	//	printf("%f\n",transferFunc1[i].x);
	//}
	//-------------------------------------------------------------------
	// create transfer function texture
  //  float4 transferFunc[] =
  //  {
  //     /* {  0.0, 0.0, 0.0, 0.0, },
  //      {  1.0, 0.0, 0.0, 1.0, },
  //      {  1.0, 0.5, 0.0, 1.0, },
  //      {  1.0, 1.0, 0.0, 1_.0, },
  //      {  0.0, 1.0, 0.0, 1.0, },
  //      {  0.0, 1.0, 1.0, 1.0, },
  //      {  0.0, 0.0, 1.0, 1.0, },
  //      {  1.0, 0.0, 1.0, 1.0, },
  //      {  0.0, 0.0, 0.0, 0.0, },*/

		//{  0.0, 0.0, 0.0, 0.0, },
  //      {  0.0, 0.0, 0.0, 1.0, },
  //      {  0.0, 0.0, 0.1, 0.2, },
  //      {  0.3, 0.4, 0.5, 0.6, },
  //      {  0.7, 0.8, 0.9, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 0.0, },
  //  };

   // create 3D array

	//hipExtent Size2 = make_hipExtent(256, 256, 1);
 //   hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
 //   checkCudaErrors(hipMalloc3DArray(&d_transferFuncArray1, &channelDesc3, Size2));

 //   // copy data to 3D array
 //   hipMemcpy3DParms copyParams3 = {0};
 //   copyParams3.srcPtr   = make_hipPitchedPtr(OTF_2D, Size2.width*sizeof(float4), Size2.width, Size2.height);
 //   copyParams3.dstArray = d_transferFuncArray1;
 //   copyParams3.extent   = Size2;
 //   copyParams3.kind     = hipMemcpyHostToDevice;
 //   checkCudaErrors(hipMemcpy3D(&copyParams3));

 //   // set texture parameters
 //   tex.normalized = true;                      // access with normalized texture coordinates
 //   tex.filterMode = hipFilterModeLinear;      // linear interpolation
 //   tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates
 //   tex.addressMode[1] = hipAddressModeBorder;
 //   // tex.addressMode[2] = hipAddressModeBorder;
 //   // bind array to 3D texture
 //   checkCudaErrors(hipBindTextureToArray(transferTex1, d_transferFuncArray1, channelDesc3));
//////////////////////////////////////////////////////////////////////////////////////////////
	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray1;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray1, &channelDesc3, sizeof(transferFunc1)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray1, 0, 0, transferFunc1, sizeof(transferFunc1), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex1, d_transferFuncArray1, channelDesc3));


	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                      brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
