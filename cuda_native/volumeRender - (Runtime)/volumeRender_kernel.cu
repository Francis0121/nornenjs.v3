#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_blockArray = 0;
hipArray *d_transferFuncArray;
hipArray *d_transferFuncArray1 = 0;
typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<VolumeType, 3, hipReadModeNormalizedFloat> tex_block;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture
texture<float4, 1, hipReadModeElementType>         transferTex1; // 1D transfer function texture
typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}
__device__ unsigned char myMAX(unsigned char a, unsigned char b)
{
	if(a >= b)
		return a;
	else 
		return b;
}
__device__ 
float3 cudaNormalize(float3 a){
	float3 temp={a.x, a.y, a.z};
	float sum = sqrt((float)(a.x*a.x + a.y*a.y + a.z*a.z));

	if(sum == 0){
		temp.x = 0;
		temp.y = 0;
		temp.z = 0;
	}else{
		temp.x /= sum;
		temp.y /= sum;
		temp.z /= sum;
	}

	return temp;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}
__device__ uchar rgbaFloatToChar(float rgba)
{
	rgba = __saturatef(rgba);   // clamp to [0.0, 1.0]
	return (uchar(rgba*255));
}
__global__ void makeBlock_kernel(unsigned char* image_p, unsigned char* dest_p, hipExtent blockSize, hipExtent volumeSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (tx >= blockSize.width || ty >= blockSize.height) return;

	for(int i=0; i<blockSize.depth; i++){
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = 0;
		unsigned char tempmax=0;

		for(int z=i*4; z<=i*4+4; z++)
			for(int y=ty*4; y<=ty*4+4; y++)
				for(int x=tx*4; x<=tx*4+4; x++){
					if(z>=volumeSize.depth || y>=volumeSize.height || x>=volumeSize.width )
						continue;
					tempmax = myMAX(tempmax, image_p[z*volumeSize.width*volumeSize.height + y*volumeSize.height + x]);
				}
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = tempmax;
	}
}
__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
	float4 temp =make_float4(0.0f);
	//uint4 sum = make_uint4(0);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d * tnear;
    float3 step = eyeRay.d*tstep;
	float max = 0.0f; 
    for (float i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates

	   // float block_den = tex3D(tex_block, (pos.x*0.5f+0.5f), (pos.y*0.5f+0.5f), (pos.z*0.5f+0.5f))*65535;
		//float3 advanced = {0.0f,0.0f.0.0f};
		//uint density = __float2uint_rn(block_den*256);
		/*temp.w = block_den;
		temp.x = block_den;
		temp.y = block_den;
		temp.z = block_den;
		uint density =  ((unsigned int)(temp.w*255)<<24) | ((unsigned int)(temp.z*255)<<16) | ((unsigned int)(temp.y*255)<<8) | (unsigned int)(temp.x*255);*/
	   //	if(block_den >= max) 
       //				max = block_den;*/
	   //if((int)block_den < 80) { //����� ���� - PALLET_START~PALLET_END������ �׸��� ������
		  // int3 nowPos= {(pos.x*0.5f+0.5f), (pos.y*0.5f+0.5f), (pos.z*0.5f+0.5f)};
		  // int3 advpos;
		  // do{
				//pos += (step*0.5);
			
		  // }
		
	    //
	    //}
		//else{
			float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
	       // float sample_next = tex3D(tex, pos.x*0.5f+0.5+(step.x*0.5), pos.y*0.5f+0.5f +(step.y*0.5), pos.z*0.5f+0.5f+(step.z*0.5));
			
			// lookup in transfer function texture
			float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
			//float4 col={0.0};
			//float diff;
			//if(sample<=sample_next){
				// diff=sample_next-sample;
				// float4 col= (tex1D(transferTex1, (sample_next-transferOffset)*transferScale) - tex1D(transferTex1, (sample-transferOffset)*transferScale)) / diff;
				
			//}
			//else if(sample>sample_next){
			//	diff=sample-sample_next;
			///	col= (tex1D(transferTex1, (sample-transferOffset)*transferScale) - tex1D(transferTex1, (sample_next-transferOffset)*transferScale)) / diff;
			//}
			//float4 col = tex3D(transferTex1,sample,sample_next,0);


			//float3 nV = {0.0, 0.0, 0.0};
			//float3 lV = {0.0, 0.0, 0.0};

			//lV.x = eyeRay.d.x;
			//lV.y = eyeRay.d.y;
			//lV.z = eyeRay.d.z;
			//
			//float x_plus = tex3D(tex, pos.x*0.5f+0.5+(step.x*0.5), pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
			//float x_minus = tex3D(tex,pos.x*0.5f+0.5-(step.x*0.5), pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

			//float y_plus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f +(step.y*0.5), pos.z*0.5f+0.5f);
			//float y_minus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f-(step.y*0.5),pos.z*0.5f+0.5f);

			//float z_plus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f+(step.z*0.5));
			//float z_minus = tex3D(tex, pos.x*0.5f+0.5, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f-(step.z*0.5));

			//nV.x = (x_plus - x_minus)/2.0f;
			//nV.y = (y_plus - y_minus)/2.0f;
			//nV.z = (z_plus - z_minus)/2.0f;

			////nV = cudaNormalize(nV);

			//float NL = 0.0f;
			//NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;

			//if(NL < 0.0f) NL = 0.0f;
			//float localShading = 0.2 + 0.8*NL;
			
			//col*=localShading;
			// pre-multiply alpha
			col.x *= col.w;
			col.y *= col.w;
			col.z *= col.w;
			// "over" operator for front-to-back blending
			sum = sum + col*(1.0f - sum.w);

			// exit early if opaque
			if (sum.w > opacityThreshold)
				break;

			t += (tstep*0.5);

			if (t > tfar) break;

			pos += (step*0.5);
		//}
	}
	/*sum.x = max;
	sum.y = max;
	sum.z = max;
	sum.w = 0;*/
    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
	
}

extern "C"
void* make_blockVolume(void* image, hipExtent blockSize, hipExtent volumeSize)
{
	unsigned int vsize = volumeSize.width * volumeSize.height * volumeSize.depth * sizeof(VolumeType);
	unsigned int bsize = blockSize.width * blockSize.height * blockSize.depth * sizeof(VolumeType);

	unsigned char *dest; //cpu�� ���� ��� data
	unsigned char *dest_p; //gpu���� ����� ��� ������
	unsigned char *image_p; //���� ������

	dest = new unsigned char[bsize/sizeof(VolumeType)]; //64*64*57
	memset((void*)dest, 0, bsize);

	hipMalloc((void**)&image_p, vsize); 
	hipMemcpy(image_p, image, vsize, hipMemcpyHostToDevice); 

	hipMalloc((void**)&dest_p, bsize);

	dim3 Db = dim3(16, 16);
	dim3 Dg = dim3(4, 4);

	makeBlock_kernel<<<Dg, Db>>>(image_p, dest_p, blockSize, volumeSize);

	hipMemcpy(dest, dest_p, bsize, hipMemcpyDeviceToHost);
	/*for(int i=0; i<64*64*47; i++)
	{
		printf("%d\n",dest[i]);
	}*/
	hipFree(image_p);
	hipFree(dest_p);

	return dest;

}
extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}
extern "C"
void initBlockTexture(void *h_volume_block, int x, int y, int z)
{
	hipExtent block_Size = make_hipExtent(x, y, z);
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors( hipMalloc3DArray(&d_blockArray, &channelDesc, block_Size) );

    // copy data to 3D array
    hipMemcpy3DParms myParams = {0};
	myParams.srcPtr   = make_hipPitchedPtr(h_volume_block, block_Size.width*sizeof(VolumeType), block_Size.width, block_Size.height);
    myParams.dstArray = d_blockArray;
    myParams.extent   = block_Size;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_block.normalized = true;                      // access with normalized texture coordinates
    tex_block.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_block.channelDesc = channelDesc;
	tex_block.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates
    tex_block.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_block, d_blockArray, channelDesc));            
} 




//struct OTF_2D* getPre_integration(){
//
//	
//	for(int x=0; x<256; x++){
//		for(int y=0; y<256; y++){
//
//			float4 result;
//			float4 temp={0.0f};
//
//			if(y > x){
//				for(int i=x; i<y; i++){
//					temp.x = transferFunc[i].x;
//					temp.y = transferFunc[i].y;
//					temp.z = transferFunc[i].z;
//					temp.w = transferFunc[i].w;
//					
//					float diff = i-x;
//
//					if(diff == 0.0f)
//						diff = 1.0f;
//
//					temp.w = 1.0f-pow(1-temp.w, 1/diff);
//
//					result.x += (1-result.w)*temp.x*temp.w;
//					result.y += (1-result.w)*temp.y*temp.w;
//					result.z += (1-result.w)*temp.z*temp.w;
//					result.w += (1-result.w)*temp.w;
//				}
//			}
//			else if(x > y){
//				for(int i=y; i<x; i++){
//					temp.x = transferFunc[i].x;
//					temp.y = transferFunc[i].y;
//					temp.z = transferFunc[i].z;
//					temp.w = transferFunc[i].w;
//
//					float diff = i-y;
//
//					if(diff == 0.0f)
//						diff = 1.0f;
//
//					temp.w = 1.0f-pow(1-temp.w, 1/diff);
//
//					result.x += (1-result.w)*temp.x*temp.w;
//					result.y += (1-result.w)*temp.y*temp.w;
//					result.z += (1-result.w)*temp.z*temp.w;
//					result.w += (1-result.w)*temp.w;
//				}
//			}
//			else {
//				result.x = 255.0f;
//				result.y = 255.0f;
//				result.z = 255.0f;
//				result.w = 0.0f;
//			}
//			OTF_2D[256*x + y].sum_R = result.x;
//			OTF_2D[256*x + y].sum_G = result.y;
//			OTF_2D[256*x + y].sum_B = result.z;
//			OTF_2D[256*x + y].sum_a = result.w;
//		}
//	}
//	return OTF_2D;
//}
extern "C"
{
	void FreeGPUVolArray(void)
	{
		hipFreeArray(d_volumeArray);
		hipFreeArray(d_blockArray);
	}
 
	void FreeGPUTFArray(void)
	{
		hipFreeArray(d_TFArray);
	}
 
	void FreeGPUEtcArray(void)
	{
		hipFreeArray(d_AverageArray);
		hipFreeArray(d_SigmaArray);
	}
}
 
 
void initTFTexture(int width, float4 *h_data)   
{
	if(d_TFArray != 0)
		hipFreeArray(d_TFArray);
 
	uint size = width*sizeof(float)*4;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	checkCudaErrors(hipMallocArray(&d_TFArray, &channelDesc, width)); 
	
	checkCudaErrors(hipMemcpyToArray(d_TFArray, 0, 0, h_data, size, hipMemcpyHostToDevice));
 
    tex_TF.addressMode[0] = hipAddressModeClamp;
    tex_TF.addressMode[1] = hipAddressModeClamp;
    tex_TF.filterMode = hipFilterModePoint;
    tex_TF.normalized = false;    // access with integer texture coordinates
	checkCudaErrors(hipBindTextureToArray(tex_TF, d_TFArray, channelDesc));
 
}
 
 
void initVolume(const ushort *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
	if(d_volumeArray != NULL) {
		hipFreeArray(d_volumeArray);
		d_volumeArray=NULL;
	}
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize, 0) );
 
    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_volumeArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );
 
    // set texture parameters
    tex_volume.normalized = false;                      // access with normalized texture coordinates
    tex_volume.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_volume.channelDesc = channelDesc;
	tex_volume.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_volume.addressMode[1] = hipAddressModeBorder;
    tex_volume.addressMode[2] = hipAddressModeBorder;
 
 
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_volume, d_volumeArray, channelDesc));
}
 
 
void initAvgVolume(const float *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_AverageArray, &channelDesc, volumeSize, 0) );
 
    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_AverageArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );
 
    // set texture parameters
    tex_average.normalized = false;                      // access with normalized texture coordinates
    tex_average.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_average.channelDesc = channelDesc;
	tex_average.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_average.addressMode[1] = hipAddressModeBorder;
    tex_average.addressMode[2] = hipAddressModeBorder;
 
 
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_average, d_AverageArray, channelDesc));
}
 
 
void initSigVolume(const float *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_SigmaArray, &channelDesc, volumeSize, 0) );
 
    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_SigmaArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );
 
    // set texture parameters
    tex_sigma.normalized = false;                      // access with normalized texture coordinates
    tex_sigma.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_sigma.channelDesc = channelDesc;
	tex_sigma.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_sigma.addressMode[1] = hipAddressModeBorder;
    tex_sigma.addressMode[2] = hipAddressModeBorder;
 
 
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_sigma, d_SigmaArray, channelDesc));
}
 
 
void initBlockTexture(const ushort *h_volume_block, hipExtent blockSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors( hipMalloc3DArray(&d_blockArray, &channelDesc, blockSize, 0) );
 
    // copy data to 3D array
	int x = blockSize.width;
	int y = blockSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume_block, x*bytePerVoxel, x, y);
    myParams.dstArray = d_blockArray;
    myParams.extent   = blockSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );
 
    // set texture parameters
    tex_block.normalized = false;                      // access with normalized texture coordinates
    tex_block.filterMode = hipFilterModePoint;      // linear interpolation
    tex_block.channelDesc = channelDesc;
	tex_block.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_block.addressMode[1] = hipAddressModeBorder;
    tex_block.addressMode[2] = hipAddressModeBorder;
 
 
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_block, d_blockArray, channelDesc));            
} 
 
 
 
__device__ void GetRayBound(float *t, float3 sdot, float3 start, hipExtent volumeSize){
 
	const float EPS = 0.00001; // epsilon
	// [0,0,0] ~ [255,255,224] box
	// eye : sdot
	// direction : start
	// get t1, t2
	float kx[2]={-20000,20000}, ky[2]={-20000,20000}, kz[2]={-20000,20000};
	// sdot.x + kx[0] * start.x = 0;
	if( fabs((float)start.x) > EPS) {
		kx[0] = (0 - sdot.x) / start.x;
		kx[1] = (volumeSize.width - sdot.x) / start.x;
		if( kx[0] > kx[1] ) { // in > out
			float temp = kx[0];
			kx[0] = kx[1];
			kx[1] = temp;
		}
	}
 
	if( fabs((float)start.y) > EPS){
		ky[0] = (0 - sdot.y) / start.y;
		ky[1] = (volumeSize.height - sdot.y) / start.y;
		if( ky[0] > ky[1] ) { // in > out
			float temp = ky[0];
			ky[0] = ky[1];
			ky[1] = temp;
		}
	}
 
	if( fabs((float)start.z) > EPS){
		kz[0] = (0 - sdot.z) / start.z;
		kz[1] = (volumeSize.depth - sdot.z) / start.z;
		if( kz[0] > kz[1] ) { // in > out
			float temp = kz[0];
			kz[0] = kz[1];
			kz[1] = temp;
		}
	}
 
	float kin = max(max(kx[0], ky[0]), kz[0]);
	float kout = min(min(kx[1], ky[1]), kz[1]);
 
	t[0] = kin + 0.05f;
	t[1] = kout - 0.05f;
 
}
 
__device__ ushort myMAX(ushort a, ushort b)
{
	if(a >= b)
		return a;
	else 
		return b;
}
 
__device__ float3 cudaNormalize(float3 a){
	float3 temp={a.x, a.y, a.z};
	float sum = sqrt((float)(a.x*a.x + a.y*a.y + a.z*a.z));
 
	if(sum == 0){
		temp.x = 0;
		temp.y = 0;
		temp.z = 0;
	}else{
		temp.x /= sum;
		temp.y /= sum;
		temp.z /= sum;
	}
 
	return temp;
}
 
 
__device__ float GetSum(float Average, float Sigma, int start, int end, float* probability_k)
{
	//start=start*16.0f;
	//end=end*16.0f;
	//Average = Average-200.0f;
	float startz = (start - Average)/Sigma;
	float endz = (end - Average)/Sigma;
	float pi = 3.141592f, e = 2.718f;
	float p1=0.0f, p2=0.0f;
 
	if(startz > 5.0f)
		p1 = 1.0f;
	else if(startz < -5.0f)
		p1 = 0.0f;
	//else if(startz < -10.0f)
	//	p1 = 0.5f;
	else if(startz >= 0.0f)
		p1 = 0.5f + probability_k[(int)(startz*100)];
	else 
		p1 = 0.5f - probability_k[-(int)(startz*100)];
 
	if(endz > 5.0f)
		p2 = 1.0f;
	else if(endz < -5.0f)
		p2 = 0.0f;
	//else if(endz < -10.0f)
	//	p2 = 0.5f;
	else if(endz >= 0.0f)
		p2 = 0.5f + probability_k[(int)(endz*100)];
	else 
		p2 = 0.5f - probability_k[-(int)(endz*100)];
 
	if(endz == startz)
		endz = startz+0.1f; //debug code
 
	
	return ((1.0f/((endz-startz)*sqrt(2.0f*pi)))*((1.0f/pow(sqrt(e), startz*startz))-
		(1.0f/pow(sqrt(e), endz*endz))) + (-startz*1.0f/(endz-startz))*(p2-p1) + 1.0f*(1-p2));
}
 
 
__global__ void makeBlock_kernel(ushort* image_p, ushort* dest_p, hipExtent blockSize, hipExtent volumeSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (tx >= blockSize.width || ty >= blockSize.height) return;
 
	for(int i=0; i<blockSize.depth; i++){
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = 0;
		ushort tempmax=0;
 
		for(int z=i*4; z<=i*4+4; z++)
			for(int y=ty*4; y<=ty*4+4; y++)
				for(int x=tx*4; x<=tx*4+4; x++){
					if(z>=volumeSize.depth || y>=volumeSize.height || x>=volumeSize.width )
						continue;
					tempmax = myMAX(tempmax, image_p[z*volumeSize.width*volumeSize.height + y*volumeSize.height + x]);
				}
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = tempmax;
	}
 
		
}
 
 
__global__ void cuda_kernel(uchar *surface, int width, int height, hipExtent volumeSize, float3 sdot, 
							float3 vDir, float3 vXcross, float3 vYcross, float zResolution, float blockResolution)
{
    int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
 
    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (tx >= width || ty >= height) return;
 
	sdot = sdot + (tx-width/2)*vXcross + (ty-height/2)*vYcross;
 
	float t[2] = {0.0f, 1000.0f};
	GetRayBound(t, sdot, vDir, volumeSize); //t1, t2�޾ƿ���
 
	float4 intensity = {0.0f};
	float alpha = 0.0f;
	bool bShading=false;
	bool bSkipping = false;
 
	for(float i=t[0]; i<t[1]; i+=1.0f){
 
		float3 render={0.0f, 0.0f, 0.0f};
		render = sdot + i*vDir;
 
		float block_den = tex3D(tex_block, (int)(render.x*blockResolution), (int)(render.y*blockResolution), 
								int(render.z*blockResolution))*65535;
		float3 advanced  = {0.0f, 0.0f, 0.0f};
		if((int)block_den < alpha_start) { 
			int3 nowPos = {(int)(render.x*blockResolution), (int)(render.y*blockResolution), 
							(int)(render.z*blockResolution)};
			int3 advPos;
			do {
				i += 1.0f;
				advanced = sdot + i*vDir;
				advPos.x = (int)(advanced.x*blockResolution);
				advPos.y = (int)(advanced.y*blockResolution);
				advPos.z = (int)(advanced.z*blockResolution);
 
			} while ( nowPos.x == advPos.x &&
					  nowPos.y == advPos.y &&
					  nowPos.z == advPos.z);
			i -= 1.0f;
			bShading=true;
			bSkipping=true;
			continue;
		}
 
		float den = tex3D(tex_volume, render.x, render.y, render.z)*65535;
		//float den_next = tex3D(tex_volume, render.x+startvec.x, render.y+startvec.y, render.z+startvec.z)*4095; //next voxel
	
		float4 samplecolor = tex1D(tex_TF, den);
		//float4 samplecolor = tex3D(tex_TF2d, den, den_next, 0); //pre-integral 
 
		if(bSkipping){
			float3 prevpos = sdot +(i-1)*vDir;
			float den_prev = tex3D(tex_volume, prevpos.x, prevpos.y, prevpos.z)*65535;
			float4 prevcolor = tex1D(tex_TF, den_prev);
		
			samplecolor +=  (1.0f-samplecolor.w)*prevcolor*prevcolor.w;
		}
		bSkipping=false;
 
		if(samplecolor.w < 0.01f) {} else
		if(samplecolor.w > 0.001f && bShading){
			//------------------------------------------------------------------------
			//shading1 - local - NL�� �̾Ƴ���
			//float shading1 = 0.0f;
			float3 nV = {0.0, 0.0, 0.0};
			float3 lV = {0.0, 0.0, 0.0};
 
			lV = vDir;
 
			float x_plus = tex3D(tex_volume, render.x+1, render.y, render.z)*65535;
			float x_minus = tex3D(tex_volume, render.x-1, render.y, render.z)*65535;
 
			float y_plus = tex3D(tex_volume, render.x, render.y+1, render.z)*65535;
			float y_minus = tex3D(tex_volume, render.x, render.y-1, render.z)*65535;
 
			float z_plus = tex3D(tex_volume, render.x, render.y, render.z+1)*65535;
			float z_minus = tex3D(tex_volume, render.x, render.y, render.z-1)*65535;
 
			nV.x = (x_plus - x_minus);
			nV.y = (y_plus - y_minus);
			nV.z = (z_plus - z_minus)*(float)zResolution;
 
			nV = cudaNormalize(nV);
 
			float NL = 0.0f;
			NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;
 
			if(NL < 0.0f) NL = 0.0f;
 
			float localShading = 0.3 + 0.7*NL;
 
			samplecolor.x *= localShading;
			samplecolor.y *= localShading;
			samplecolor.z *= localShading;
		} else
		{
			const float fCutPlaneShading = 0.0f;
			samplecolor = samplecolor*fCutPlaneShading;
		
		}
		bShading = true;
 
		intensity.x += (1.0f-alpha)*samplecolor.x*samplecolor.w;
		intensity.y += (1.0f-alpha)*samplecolor.y*samplecolor.w;
		intensity.z += (1.0f-alpha)*samplecolor.z*samplecolor.w;
		alpha += (1.0f-alpha)*samplecolor.w;
 
		if(alpha > 0.95f)
			break;
 
	}
 
	surface[(ty*width + tx)*3 + 0] = intensity.x;
	surface[(ty*width + tx)*3 + 1] = intensity.y;
	surface[(ty*width + tx)*3 + 2] = intensity.z;
}
 
__global__ void cuda_kernel_AO(uchar *surface, int width, int height, hipExtent volumeSize, float3 sdot, 
							float3 vDir, float3 vXcross, float3 vYcross, float zResolution, float blockResolution,
							float* probability_k, float3 factor)
{
    int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
 
    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (tx >= width || ty >= height) return;
 
	sdot = sdot + (tx-width/2)*vXcross + (ty-height/2)*vYcross;
 
	float t[2] = {0.0f, 1000.0f};
	GetRayBound(t, sdot, vDir, volumeSize); //t1, t2�޾ƿ���
 
	float4 intensity = {0.0f};
	float alpha = 0.0f;
	bool bShading=false;
	bool bSkipping = false;
 
	for(float i=t[0]; i<t[1]; i+=1.0f){
 
		float3 render={0.0f, 0.0f, 0.0f};
		render = sdot + i*vDir;
 
		float block_den = tex3D(tex_block, (int)(render.x*blockResolution), (int)(render.y*blockResolution), 
								int(render.z*blockResolution))*65535;
		float3 advanced  = {0.0f, 0.0f, 0.0f};
		if((int)block_den < alpha_start) { 
			int3 nowPos = {(int)(render.x*blockResolution), (int)(render.y*blockResolution), 
							(int)(render.z*blockResolution)};
			int3 advPos;
			do {
				i += 1.0f;
				advanced = sdot + i*vDir;
				advPos.x = (int)(advanced.x*blockResolution);
				advPos.y = (int)(advanced.y*blockResolution);
				advPos.z = (int)(advanced.z*blockResolution);
 
			} while ( nowPos.x == advPos.x &&
					  nowPos.y == advPos.y &&
					  nowPos.z == advPos.z);
			i -= 1.0f;
			bShading=true;
			bSkipping=true;
			continue;
		}
 
		float den = tex3D(tex_volume, render.x, render.y, render.z)*65535;
		//float den_next = tex3D(tex_volume, render.x+startvec.x, render.y+startvec.y, render.z+startvec.z)*4095; //next voxel
	
		float4 samplecolor = tex1D(tex_TF, den);
		//float4 samplecolor = tex3D(tex_TF2d, den, den_next, 0); //pre-integral 
 
		if(bSkipping){
			float3 prevpos = sdot +(i-1)*vDir;
			float den_prev = tex3D(tex_volume, prevpos.x, prevpos.y, prevpos.z)*65535;
			float4 prevcolor = tex1D(tex_TF, den_prev);
		
			samplecolor +=  (1.0f-samplecolor.w)*prevcolor*prevcolor.w;
		}
		bSkipping=false;
 
		if(samplecolor.w < 0.01f) {} else
		if(samplecolor.w > 0.001f && bShading){
			//------------------------------------------------------------------------
			//shading1 - local - NL�� �̾Ƴ���
			//float shading1 = 0.0f;
			float3 nV = {0.0, 0.0, 0.0};
			float3 lV = {0.0, 0.0, 0.0};
 
			lV = vDir;
 
			float x_plus = tex3D(tex_volume, render.x+1, render.y, render.z)*65535;
			float x_minus = tex3D(tex_volume, render.x-1, render.y, render.z)*65535;
 
			float y_plus = tex3D(tex_volume, render.x, render.y+1, render.z)*65535;
			float y_minus = tex3D(tex_volume, render.x, render.y-1, render.z)*65535;
 
			float z_plus = tex3D(tex_volume, render.x, render.y, render.z+1)*65535;
			float z_minus = tex3D(tex_volume, render.x, render.y, render.z-1)*65535;
 
			nV.x = (x_plus - x_minus);
			nV.y = (y_plus - y_minus);
			nV.z = (z_plus - z_minus)*(float)zResolution;
 
			nV = cudaNormalize(nV);
 
			float NL = 0.0f;
			NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;
 
			if(NL < 0.0f) NL = 0.0f;
 
			float localShading = 0.3 + 0.7*NL;
			//------------------------------------------------------------------------
			//shading2 - global
			nV *= 5.f;
			float Sigma = tex3D(tex_sigma, min(render.x-nV.x, (float)volumeSize.width), 
				min(render.y-nV.y, (float)volumeSize.height), min(render.z-nV.z, (float)volumeSize.depth));
			float Average = tex3D(tex_average, min(render.x-nV.x, (float)volumeSize.width), 
				min(render.y-nV.y, (float)volumeSize.height), min(render.z-nV.z, (float)volumeSize.depth));	
 
			//samplecolor = tex1D(tex_TF, Average);
 
			float sum = GetSum(Average, Sigma, alpha_start, alpha_end, probability_k); //1400, 2100 - alpha starat, end
 
			//if(x_plus > den && y_plus > den && z_plus > den )
			//	sum = sum/1.5f;
 
			float shading2 = 1.0f - min(max((sum*2.0f - 0.5f), 0.0f), 1.0f); //global shding value ����
			//shading2 = 1.0f-shading2;
 
			float shading = factor.x + factor.y*shading2*shading2 + factor.z*NL; //factor1,2,3
 
			samplecolor.x *= shading2;
			samplecolor.y *= shading2;
			samplecolor.z *= shading2;
		} else
		{
			const float fCutPlaneShading = 0.0f;
			samplecolor = samplecolor*fCutPlaneShading;
		
		}
		bShading = true;
 
		intensity.x += (1.0f-alpha)*samplecolor.x*samplecolor.w;
		intensity.y += (1.0f-alpha)*samplecolor.y*samplecolor.w;
		intensity.z += (1.0f-alpha)*samplecolor.z*samplecolor.w;
		alpha += (1.0f-alpha)*samplecolor.w;
 
		if(alpha > 0.95f)
			break;
 
	}
 
	surface[(ty*width + tx)*3 + 0] = intensity.x;
	surface[(ty*width + tx)*3 + 1] = intensity.y;
	surface[(ty*width + tx)*3 + 2] = intensity.z;
}
 
ushort* make_blockVolume(ushort* image, hipExtent blockSize, hipExtent volumeSize)
{
	unsigned int vsize = volumeSize.width * volumeSize.height * volumeSize.depth * sizeof(ushort);
	unsigned int bsize = blockSize.width * blockSize.height * blockSize.depth * sizeof(ushort);
 
	ushort *dest, *dest_p, *image_p;
 
	dest = new ushort[bsize/sizeof(ushort)];
	memset((void*)dest, 0, bsize);
 
	hipMalloc((void**)&image_p, vsize);
	hipMemcpy(image_p, image, vsize, hipMemcpyHostToDevice);
 
	hipMalloc((void**)&dest_p, bsize);
 
	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((blockSize.width+Db.x-1)/Db.x, (blockSize.height+Db.y-1)/Db.y);
 
	makeBlock_kernel<<<Dg, Db>>>(image_p, dest_p, blockSize, volumeSize);
 
	hipMemcpy(dest, dest_p, bsize, hipMemcpyDeviceToHost);
 
	hipFree(image_p);
	hipFree(dest_p);
 
	return dest;
 
}
 
 
void Run_Kernel(uchar* surface, const int imgsize[2], hipExtent volumeSize, ushort* pVol,
				float zResolution, float blockResolution, const float *ViewingPoint)
{
	printf("-GPU render : Basic\n");
	//---------------------------------------------------------------
	//����, ī�޶� ���� ����
	float3 volCenter = {volumeSize.width/2.0f, volumeSize.height/2.0f, volumeSize.depth/2.0f};
	float3 sdot={ViewingPoint[0], ViewingPoint[1], ViewingPoint[2]}, vUp={0.0f, 0.0f, 1.0f};
	float3 frontView = {volumeSize.width/2.f, volumeSize.height, volumeSize.depth/2.f};
	
	float3 vDir, vXCross, vYcross, front;
 
	front = frontView-volCenter;
	front = normalize(front);
 
	vDir = volCenter-sdot;
	vDir = normalize(vDir);
 
	float3 temp_z = {0.f, vDir.y, vDir.z};
	temp_z = normalize(temp_z);
	if(dot(front, temp_z) < 0.f)
		vUp.z = -1.0f;
	
	vXCross = cross(vUp, vDir);
	vXCross = normalize(vXCross);
 
	vYcross = cross(vDir, vXCross);
	vYcross = normalize(vYcross);
	//---------------------------------------------------------------
 
	uchar* surface_k;
	hipMalloc((void**)&surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	hipMemset(surface_k, 0, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	hipMemcpy(surface_k, surface, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyHostToDevice);
 
    //dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((imgsize[0]+Db.x-1)/Db.x, (imgsize[1]+Db.y-1)/Db.y);
 
    cuda_kernel<<<Dg,Db>>>(surface_k, imgsize[0], imgsize[1], volumeSize, sdot, vDir, 
		vXCross, vYcross, zResolution, blockResolution);
    if (hipGetLastError() != hipSuccess)
        printf("cuda_kernel() failed to launch error = %d\n", hipGetLastError());    
 
	hipMemcpy(surface, surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyDeviceToHost);
	hipFree(surface_k);
}
 
 
void Run_Kernel_AO(uchar* surface, const int imgsize[2], hipExtent volumeSize, ushort* pVol,
				float zResolution, float blockResolution, float probability[310], float factor[3], const float *ViewingPoint)
{
	printf("-GPU render : AO \n");
	//---------------------------------------------------------------
	//����, ī�޶� ���� ����
	float3 volCenter = {volumeSize.width/2.0f, volumeSize.height/2.0f, volumeSize.depth/2.0f};
	float3 sdot={ViewingPoint[0], ViewingPoint[1], ViewingPoint[2]}, vUp={0.0f, 0.0f, 1.0f};
	float3 frontView = {volumeSize.width/2.f, volumeSize.height, volumeSize.depth/2.f};
	
	float3 vDir, vXCross, vYcross, front;
 
	front = frontView-volCenter;
	front = normalize(front);
 
	vDir = volCenter-sdot;
	vDir = normalize(vDir);
 
	float3 temp_z = {0.f, vDir.y, vDir.z};
	temp_z = normalize(temp_z);
	if(dot(front, temp_z) < 0.f)
		vUp.z = -1.0f;
	
	vXCross = cross(vUp, vDir);
	vXCross = normalize(vXCross);
 
	vYcross = cross(vDir, vXCross);
	vYcross = normalize(vYcross);
	//---------------------------------------------------------------
 
	float* probability_k;
	hipMalloc((void**)&probability_k, 310*sizeof(float));
	hipMemset(probability_k, 0, 310*sizeof(float));
	hipMemcpy(probability_k, probability, 310*sizeof(float), hipMemcpyHostToDevice);
	
	float3 factor3 ={factor[0], factor[1], factor[2]};
 
	uchar* surface_k;
	hipMalloc((void**)&surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	hipMemset(surface_k, 0, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	hipMemcpy(surface_k, surface, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyHostToDevice);
 
    //dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((imgsize[0]+Db.x-1)/Db.x, (imgsize[1]+Db.y-1)/Db.y);
 
    cuda_kernel_AO<<<Dg,Db>>>(surface_k, imgsize[0], imgsize[1], volumeSize, sdot, vDir, 
		vXCross, vYcross, zResolution, blockResolution, probability_k, factor3);
 
    if (hipGetLastError() != hipSuccess)
        printf("cuda_kernel() failed to launch error = %d\n", hipGetLastError());
    
	hipMemcpy(surface, surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyDeviceToHost);
	hipFree(surface_k);
}
 
 
void initTF2dTexture(float4 *h_volume, int x, int y, int z)
{
	hipExtent Size = make_hipExtent(x, y, z);
    // create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_TF2dArray, &channelDesc, Size, 0) );
 
    // copy data to 3D array
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*sizeof(float4), x, y);
    myParams.dstArray = d_TF2dArray;
    myParams.extent   = Size;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );
 
    // set texture parameters
    tex_TF2d.normalized = false;                      // access with normalized texture coordinates
    tex_TF2d.filterMode = hipFilterModePoint;      // linear interpolation
    tex_TF2d.channelDesc = channelDesc;
	tex_TF2d.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_TF2d.addressMode[1] = hipAddressModeBorder;
    tex_TF2d.addressMode[2] = hipAddressModeBorder;
 
 
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_TF2d, d_TF2dArray, channelDesc));
}
 
 
 
__global__ void TF2d_kernel(float4* TF2d_k, int TFSize)
{
	int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
 
	if(x>=TFSize || y>=TFSize)
		return;
 
	//float4 result;				//1�� ��� - pre-integral : OTF �����ϰ� �ص� �Ѱ㸸 ������ �Ҽ��ִ�.
	//float4 temp = {0.0f};
	//
	//if(y > x){
	//	for(int i=x; i<y; i++){
	//		temp = tex1D(tex_TF, i);
 
	//		float diff = i-x;
 
	//		if(diff == 0.0f)
	//			diff = 1.0f;
 
	//		temp.w = 1.0f-pow(1-temp.w, 1/diff);
 
	//		result.x += (1-result.w)*temp.x*temp.w;
	//		result.y += (1-result.w)*temp.y*temp.w;
	//		result.z += (1-result.w)*temp.z*temp.w;
	//		result.w += (1-result.w)*temp.w;
	//	}
	//}
	//else if(x > y){
	//	for(int i=y; i<x; i++){
	//		temp = tex1D(tex_TF, i);
 
	//		float diff = i-y;
 
	//		if(diff == 0.0f)
	//			diff = 1.0f;
 
	//		temp.w = 1.0f-pow(1-temp.w, 1/diff);
 
	//		result.x += (1-result.w)*temp.x*temp.w;
	//		result.y += (1-result.w)*temp.y*temp.w;
	//		result.z += (1-result.w)*temp.z*temp.w;
	//		result.w += (1-result.w)*temp.w;
	//	}
	//}
	//else {
	//	result.x = 255.0f;
	//	result.y = 255.0f;
	//	result.z = 255.0f;
	//	result.w = 0.0f;
	//}
 
	float4 temp;					//2�� ��� - 1��������� ���ṫ�̰� �� ����� : summed 2d table
	float4 result = {0.0};
	float4 sum = {0.0f};
	
	int nx, ny, diff;
	if(x>y){
		diff = x-y;
		ny = x;
		nx = y;
	}
	else if(y>x){
		diff = y-x;
		nx = x;
		ny = y;
	}
	else{
		diff=1;
		nx = ny = x;
		sum.w = 0.0f;
	}
 
	for(int i=nx; i<ny; i++){
		temp = tex1D(tex_TF, i);
 
		temp.x *= temp.w;
		temp.y *= temp.w;
		temp.z *= temp.w;
 
		sum.x += temp.x;
		sum.y += temp.y;
		sum.z += temp.z;
		sum.w += temp.w;
	}
 
	result.x = sum.x / diff; //* (newAlpha/sum.w);
	result.y = sum.y / diff; //* (newAlpha/sum.w);
	result.z = sum.z / diff; //* (newAlpha/sum.w);
	result.w = sum.w / diff;
 
		
 
	TF2d_k[TFSize*y + x].x = result.x;
	TF2d_k[TFSize*y + x].y = result.y;
	TF2d_k[TFSize*y + x].z = result.z;
	TF2d_k[TFSize*y + x].w = result.w;
 
 
}
 
 
void init_TF2d(int TFSize)
{
	int size = TFSize*TFSize;
	float4* TF2d_k;
	hipMalloc((void**)&TF2d_k, size*sizeof(float4));
	hipMemset(TF2d_k, 0, size*sizeof(float4));
 
	dim3 Db = dim3( 16, 16 ); 
    dim3 Dg = dim3( 256, 256 );
 
 
	TF2d_kernel<<<Dg, Db>>>(TF2d_k, TFSize); //pre-integral OTF init kernel - threads 4096*4096
 
	float4* TF2d;
	TF2d = new float4[size];
	memset(TF2d, 0, size*sizeof(float4));
 
	hipMemcpy(TF2d, TF2d_k, size*sizeof(float4), hipMemcpyDeviceToHost);
 
	hipFree(TF2d_k);
 
	initTF2dTexture(TF2d, TFSize, TFSize, 1);
 
	delete[] TF2d;
 
 
}
 
extern "C"
void GPU_Render(uchar *image, int imgsize[2], ushort* pVol, int dim[3], 
				TF *transfer, int tf_size, double zResolution, bool &bInitVol, bool &bInitTF, float *ViewingPoint)
{
	float4 *tf_cuda;
	if(!bInitTF){
		printf("-init TF texture memory - GPU\n");
		tf_cuda = new float4[tf_size];
		for(int i=0; i<tf_size; i++){
			tf_cuda[i].x = transfer[i].R;
			tf_cuda[i].y = transfer[i].G;
			tf_cuda[i].z = transfer[i].B;
			tf_cuda[i].w = transfer[i].alpha;
		}
		initTFTexture(tf_size, tf_cuda);
	}
 
	hipExtent volume_dim_block, volume_dim;
	float blockResolution = 0.25f;
	volume_dim = make_hipExtent(dim[0], dim[1], dim[2]);
	volume_dim_block = make_hipExtent(dim[0]*blockResolution, dim[1]*blockResolution, dim[2]*blockResolution);
 
	ushort *pVol_block;
	if(!bInitVol){
		printf("-init Volume texture memory - GPU\n");
		pVol_block = make_blockVolume(pVol, volume_dim_block, volume_dim);
 
		initVolume(pVol, volume_dim , sizeof(ushort));
		initBlockTexture(pVol_block, volume_dim_block, sizeof(ushort));
	}
 
	Run_Kernel(image, imgsize, volume_dim, pVol, (float)zResolution, blockResolution, ViewingPoint);
 
	if(!bInitVol){
		delete[] pVol_block;
		bInitVol = true;
	}
	if(!bInitTF){
		delete[] tf_cuda;
		bInitTF = true;
	}
	
}
 
extern "C"
void GPU_Render_AO(uchar *image, int imgsize[2], ushort* pVol, int dim[3], 
				TF *transfer, int tf_size, double zResolution, bool &bInitVol, bool &bInitTF,
				float *Avg, float *Sig, bool &m_bInitAvgSig, float probability[310], float factor[3],
				float *ViewingPoint)
{
	float4 *tf_cuda;
	if(!bInitTF){
		printf("-init TF texture memory - GPU\n");
		tf_cuda = new float4[tf_size];
		for(int i=0; i<tf_size; i++){
			tf_cuda[i].x = transfer[i].R;
			tf_cuda[i].y = transfer[i].G;
			tf_cuda[i].z = transfer[i].B;
			tf_cuda[i].w = transfer[i].alpha;
		}
		initTFTexture(tf_size, tf_cuda);
	}
 
	hipExtent volume_dim_block, volume_dim;
	float blockResolution = 0.25f;
	volume_dim = make_hipExtent(dim[0], dim[1], dim[2]);
	volume_dim_block = make_hipExtent(dim[0]*blockResolution, dim[1]*blockResolution, dim[2]*blockResolution);
 
	ushort *pVol_block;
	if(!bInitVol){
		printf("-init Volume texture memory - GPU\n");
		pVol_block = make_blockVolume(pVol, volume_dim_block, volume_dim);
 
		initVolume(pVol, volume_dim , sizeof(ushort));
		initBlockTexture(pVol_block, volume_dim_block, sizeof(ushort));
	}
	if(!m_bInitAvgSig && Avg != NULL && Sig != NULL){
		initAvgVolume(Avg, volume_dim, sizeof(float));
		initSigVolume(Sig, volume_dim, sizeof(float));
	}
 
	Run_Kernel_AO(image, imgsize, volume_dim, pVol, (float)zResolution, blockResolution, probability, factor, ViewingPoint);
 
	if(!bInitVol){
		delete[] pVol_block;
		bInitVol = true;
	}
	if(!bInitTF){
		delete[] tf_cuda;
		bInitTF = true;
	}	
}
 
__global__ void cuda_kernel_test(ushort *new_vol_k, ushort *vol_k, int3 dim3, float *gaussianMask_k, int maskSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
 
	if (tx >= dim3.x-2 || ty >= dim3.y-2) return;
	if (tx <= 2 || ty <= 2) return;
 
	int size = maskSize/2;
 
	for(int tz=1; tz<dim3.z; tz++)
	{
		double sum = 0.f;
		for(int i=-size; i<size+1; i++)
		{
			for(int j=-size; j<size+1; j++)
			{
				for(int k=-size; k<size+1; k++)
				{
					int z=k+1, y=j+1, x=i+1;
					sum += vol_k[(tz+k)*dim3.x*dim3.y + (ty+j)*dim3.y + (tx+i)]*gaussianMask_k[z*maskSize*maskSize + y*maskSize + x];
				}
			}
		}
		new_vol_k[tz*dim3.x*dim3.y + ty*dim3.y + tx] = (ushort)sum;
	}
 
}
 
extern "C"
void testSmoothFilter(ushort* pVol, int *dim)
{
	printf("-GPU testSmoothFilter \n");
 
	float fSigma=0.8f;
	float gaussianMask[27];
	int maskSize = 3;
	int allocSize= maskSize*maskSize*maskSize;
 
	float sum=0.f;
	for(int i=0; i<maskSize; i++)
	{
		float z = fabs((float)i-1.f);
		for(int j=0; j<maskSize; j++)
		{
			float y = fabs((float)j-1.f);
			for(int k=0; k<maskSize; k++)
			{
				float x = fabs((float)k-1.f); 
				float fDist = x+y+z;
				sum += gaussianMask[k*maskSize*maskSize + j*maskSize + i] = 
					exp(-(fDist*fDist)/(2.f*fSigma*fSigma))/(sqrtf(2.f*PI)*fSigma);
			}
		}
	}
	for(int i=0; i<maskSize; i++)
	{
		for(int j=0; j<maskSize; j++)
		{
			for(int k=0; k<maskSize; k++)
			{
				gaussianMask[k*maskSize*maskSize + j*maskSize + i] /= sum;
			}
		}
	}
 
	//printf("%f %f %f\n", gaussianMask[0], gaussianMask[1], gaussianMask[2]);
	//printf("%f %f %f\n", gaussianMask[3], gaussianMask[4], gaussianMask[5]);
	//printf("%f %f %f\n\n", gaussianMask[6], gaussianMask[7], gaussianMask[8]);
 
	//printf("%f %f %f\n", gaussianMask[9], gaussianMask[10], gaussianMask[11]);
	//printf("%f %f %f\n", gaussianMask[12], gaussianMask[13], gaussianMask[14]);
	//printf("%f %f %f\n\n", gaussianMask[15], gaussianMask[16], gaussianMask[17]);
 
	//printf("%f %f %f\n", gaussianMask[18], gaussianMask[19], gaussianMask[20]);
	//printf("%f %f %f\n", gaussianMask[21], gaussianMask[22], gaussianMask[23]);
	//printf("%f %f %f\n\n", gaussianMask[24], gaussianMask[25], gaussianMask[26]);
 
	float* gaussianMask_k;
	hipMalloc((void**)&gaussianMask_k, allocSize*sizeof(float));
	hipMemset(gaussianMask_k, 0, allocSize*sizeof(float));
	hipMemcpy(gaussianMask_k, gaussianMask, allocSize*sizeof(float), hipMemcpyHostToDevice);
 
	ushort *pVol_k, *new_pVol_k;
	int vol_size = dim[0]*dim[1]*dim[2];
	int3 vol_dim3 = {dim[0], dim[1], dim[2]};
 
	hipMalloc((void**)&pVol_k, vol_size*sizeof(ushort));
	hipMemset(pVol_k, 0, vol_size*sizeof(ushort));
	hipMemcpy(pVol_k, pVol, vol_size*sizeof(ushort), hipMemcpyHostToDevice);
 
	hipMalloc((void**)&new_pVol_k, vol_size*sizeof(ushort));
	hipMemset(new_pVol_k, 0, vol_size*sizeof(ushort));
 
	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((dim[0]+Db.x-1)/Db.x, (dim[1]+Db.y-1)/Db.y);
 
    cuda_kernel_test<<<Dg,Db>>>(new_pVol_k, pVol_k, vol_dim3, gaussianMask_k, maskSize);
 
    if (hipGetLastError() != hipSuccess)
        printf("cuda_kernel() failed to launch error = %d\n", hipGetLastError());
    
	memset(pVol, 0, sizeof(ushort)*vol_size);
	hipMemcpy(pVol, new_pVol_k, vol_size*sizeof(ushort), hipMemcpyDeviceToHost);
 
	hipFree(pVol_k);
	hipFree(new_pVol_k);
	hipFree(gaussianMask_k);
 
}
extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeBorder;
    // tex.addressMode[2] = hipAddressModeBorder;
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

		
	float4 transferFunc[256];
	float4 transferFunc1[256]={0.0f};
	 for(int i=0; i<=80; i++){    //alpha
		 transferFunc[i].w = 0.0f;
		 transferFunc[i].x = 0.0f;
		 transferFunc[i].y = 0.0f;
		 transferFunc[i].z = 0.0f;
	}
	for(int i=80+1; i<=100; i++){
		transferFunc[i].w = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].x = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].y = (1.0 / (100-80)) * ( i - 80);
		transferFunc[i].z = (1.0 / (100-80)) * ( i - 80);
	}
	for(int i=100+1; i<256; i++){
		transferFunc[i].w =1.0f;
		transferFunc[i].x =1.0f;
		transferFunc[i].y =1.0f;
		transferFunc[i].z =1.0f;
	}
	
	transferFunc1[0].w= transferFunc[0].w;
	transferFunc1[0].x= transferFunc[0].x * transferFunc[0].w;
	transferFunc1[0].y= transferFunc[0].y * transferFunc[0].w;
	transferFunc1[0].z= transferFunc[0].z * transferFunc[0].w;		
	
	for(int i=1; i<256; i++)
	{		
		

		transferFunc1[i].w += transferFunc1[i-1].w + transferFunc[i].w;
		transferFunc1[i].x += transferFunc1[i-1].x + transferFunc[i].x * transferFunc[i].w;
		transferFunc1[i].y += transferFunc1[i-1].y + transferFunc[i].y * transferFunc[i].w;
		transferFunc1[i].z += transferFunc1[i-1].z + transferFunc[i].z * transferFunc[i].w;
		
		transferFunc1[i].w =(transferFunc1[i].w/256.0f);
		transferFunc1[i].x =(transferFunc1[i].x/256.0f);
		transferFunc1[i].y =(transferFunc1[i].y/256.0f);
		transferFunc1[i].z =(transferFunc1[i].z/256.0f);
		//printf("%f %f\n",transferFunc1[i].w/256,transferFunc1[i].x/256);
		//printf("%f,%f,%f,%f\n",tempA[i],OTF_2Da[before],tempG[i],OTF_2Dg[before]);

	}
	//for(int x=0; x<256; x++){
	//	for(int y=0; y<256; y++){

	//		float4 result;
	//		float4 temp={0.0f};

	//		if(y > x){
	//			for(int i=x; i<y; i++){
	//				temp.x = transferFunc[i].x;
	//				temp.y = transferFunc[i].y;
	//				temp.z = transferFunc[i].z;
	//				temp.w = transferFunc[i].w;
	//				
	//				float diff = i-x;

	//				if(diff == 0.0f)
	//					diff = 1.0f;

	//				temp.w = 1.0f-pow(1-temp.w, 1/diff);

	//				result.x += (1-result.w)*temp.x*temp.w;
	//				result.y += (1-result.w)*temp.y*temp.w;
	//				result.z += (1-result.w)*temp.z*temp.w;
	//				result.w += (1-result.w)*temp.w;
	//			}
	//		}
	//		else if(x > y){
	//			for(int i=y; i<x; i++){
	//				temp.x = transferFunc[i].x;
	//				temp.y = transferFunc[i].y;
	//				temp.z = transferFunc[i].z;
	//				temp.w = transferFunc[i].w;

	//				float diff = i-y;

	//				if(diff == 0.0f)
	//					diff = 1.0f;

	//				temp.w = 1.0f-pow(1-temp.w, 1/diff);

	//				result.x += (1-result.w)*temp.x*temp.w;
	//				result.y += (1-result.w)*temp.y*temp.w;
	//				result.z += (1-result.w)*temp.z*temp.w;
	//				result.w += (1-result.w)*temp.w;
	//			}
	//		}
	//		else {
	//			result.x = 1.0f;
	//			result.y = 1.0f;
	//			result.z = 1.0f;
	//			result.w = 0.0f;
	//		}
	//		OTF_2D[256*x + y].sum_R = result.x;
	//		OTF_2D[256*x + y].sum_G = result.y;
	//		OTF_2D[256*x + y].sum_B = result.z;
	//		OTF_2D[256*x + y].sum_a = result.w;
	//	}
	//}
	//struct OTF_2D *p;
	//p=getPre_integration();
	//for(int i=0; i<256; i++)
	//{
	//	printf("%f\n",transferFunc1[i].x);
	//}
	//-------------------------------------------------------------------
	// create transfer function texture
  //  float4 transferFunc[] =
  //  {
  //     /* {  0.0, 0.0, 0.0, 0.0, },
  //      {  1.0, 0.0, 0.0, 1.0, },
  //      {  1.0, 0.5, 0.0, 1.0, },
  //      {  1.0, 1.0, 0.0, 1_.0, },
  //      {  0.0, 1.0, 0.0, 1.0, },
  //      {  0.0, 1.0, 1.0, 1.0, },
  //      {  0.0, 0.0, 1.0, 1.0, },
  //      {  1.0, 0.0, 1.0, 1.0, },
  //      {  0.0, 0.0, 0.0, 0.0, },*/

		//{  0.0, 0.0, 0.0, 0.0, },
  //      {  0.0, 0.0, 0.0, 1.0, },
  //      {  0.0, 0.0, 0.1, 0.2, },
  //      {  0.3, 0.4, 0.5, 0.6, },
  //      {  0.7, 0.8, 0.9, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 1.0, },
  //      {  1.0, 1.0, 1.0, 0.0, },
  //  };

   // create 3D array

	//hipExtent Size2 = make_hipExtent(256, 256, 1);
 //   hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
 //   checkCudaErrors(hipMalloc3DArray(&d_transferFuncArray1, &channelDesc3, Size2));

 //   // copy data to 3D array
 //   hipMemcpy3DParms copyParams3 = {0};
 //   copyParams3.srcPtr   = make_hipPitchedPtr(OTF_2D, Size2.width*sizeof(float4), Size2.width, Size2.height);
 //   copyParams3.dstArray = d_transferFuncArray1;
 //   copyParams3.extent   = Size2;
 //   copyParams3.kind     = hipMemcpyHostToDevice;
 //   checkCudaErrors(hipMemcpy3D(&copyParams3));

 //   // set texture parameters
 //   tex.normalized = true;                      // access with normalized texture coordinates
 //   tex.filterMode = hipFilterModeLinear;      // linear interpolation
 //   tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates
 //   tex.addressMode[1] = hipAddressModeBorder;
 //   // tex.addressMode[2] = hipAddressModeBorder;
 //   // bind array to 3D texture
 //   checkCudaErrors(hipBindTextureToArray(transferTex1, d_transferFuncArray1, channelDesc3));
//////////////////////////////////////////////////////////////////////////////////////////////
	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray1;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray1, &channelDesc3, sizeof(transferFunc1)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray1, 0, 0, transferFunc1, sizeof(transferFunc1), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex1, d_transferFuncArray1, channelDesc3));


	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                      brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
